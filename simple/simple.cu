#include <stdio.h>
#include <hip/hip_runtime.h>

//__device__ float getSin(float f2)
//{
//  return __sin(f2);
//}

// Kernel that executes on the CUDA device
__global__ void square_array(float *a, int N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx<N) a[idx] = a[idx] * a[idx];
}

__global__ void sin_array(float *a, int N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx<N) a[idx] = __sinf(a[idx]);
}

__global__ void cos_array(float *a, int N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx<N) a[idx] = __cosf(a[idx]);
}

void do_sin() {
  printf("[info] start sin\n");
  float *a_h, *a_d;  // Pointer to host & device arrays
  const int N = 10;  // Number of elements in arrays
  size_t size = N * sizeof(float);
  a_h = (float *)malloc(size);        // Allocate array on host
  hipMalloc((void **) &a_d, size);   // Allocate array on device

  printf("[info] prepare data\n");
  // Initialize host array and copy it to CUDA device
  for (int i=0; i<N; i++) a_h[i] = (float)i;
  // Print original results
  for (int i=0; i<N; i++) printf("%d %f\n", i, a_h[i]);

  hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
  // Do calculation on device:
  int block_size = 4;
  int n_blocks = N/block_size + (N%block_size == 0 ? 0:1);
  //square_array <<< n_blocks, block_size >>> (a_d, N);
  sin_array <<< n_blocks, block_size >>> (a_d, N);
  // Retrieve result from device and store it in host array
  hipMemcpy(a_h, a_d, sizeof(float)*N, hipMemcpyDeviceToHost);

  printf("[info] result\n");
  // Print results
  for (int i=0; i<N; i++) printf("%d %f\n", i, a_h[i]);
  // Cleanup
  free(a_h); hipFree(a_d);

  printf("[info] exit sin\n");
}

void do_cos() {
  printf("[info] start cos\n");
  float *a_h, *a_d;  // Pointer to host & device arrays
  const int N = 10;  // Number of elements in arrays
  size_t size = N * sizeof(float);
  a_h = (float *)malloc(size);        // Allocate array on host
  hipMalloc((void **) &a_d, size);   // Allocate array on device

  printf("[info] prepare data\n");
  // Initialize host array and copy it to CUDA device
  for (int i=0; i<N; i++) a_h[i] = (float)i;
  // Print original results
  for (int i=0; i<N; i++) printf("%d %f\n", i, a_h[i]);

  hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
  // Do calculation on device:
  int block_size = 4;
  int n_blocks = N/block_size + (N%block_size == 0 ? 0:1);
  //square_array <<< n_blocks, block_size >>> (a_d, N);
  cos_array <<< n_blocks, block_size >>> (a_d, N);
  // Retrieve result from device and store it in host array
  hipMemcpy(a_h, a_d, sizeof(float)*N, hipMemcpyDeviceToHost);

  printf("[info] result\n");
  // Print results
  for (int i=0; i<N; i++) printf("%d %f\n", i, a_h[i]);
  // Cleanup
  free(a_h); hipFree(a_d);

  printf("[info] exit cos\n");
}
 
// main routine that executes on the host
int main(void)
{
  printf("[info] start main\n");
  do_sin();
  do_cos();
  printf("[info] exit main\n");
}
